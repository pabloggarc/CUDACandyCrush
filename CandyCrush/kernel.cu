#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

int N = 10; 
int M = 5; 
int dificultad; 
int modo; 

/*__global__ void encontrar_caminos(char* tablero, int N, int M, int fila, int columna) {
    int selec = fila * N + columna; 
    int id = threadIdx.y * N + threadIdx.x; 

    //Funcion que busque camino
    int* camino = (int*)malloc(N * M * sizeof(int)); 
    int* visitados = (int*)malloc(N * M * sizeof(int));

    for (int i = 0; i < N * M; ++i) {
        camino[i] = -1; 
        visitados[i] = -1; 
    }

    buscar_camino(tablero, id, selec, visitados, 0, camino, 0); 

    printf("Camino desde %d\n: ", id); 
    for (int i = 0; i < N * M; ++i) {
        printf("%d, ", camino[i]); 
    }
}*/

void cargar_argumentos(int argc, char* argv[]) {
    if (argc != 5) {
        perror("Se esperaban argumentos -a/-m 1/2 n m. "); 
        exit(-1); 
    }
    else {
        int error = 0; 
        if (!strcmp(argv[1], "-a")) {
            modo = 0; 
        }
        else if (!strcmp(argv[1], "-m")){
            modo = 1; 
        }
        else {
            error = 1; 
        }

        if (!strcmp(argv[2], "1")) {
            dificultad = 0; 
        }
        else if (!strcmp(argv[2], "2")) {
            dificultad = 1; 
        }
        else {
            error = 1; 
        }

        if (atoi(argv[3]) < 1 || atoi(argv[4]) < 1) {
            error = 1; 
        }
        else {
            N = atoi(argv[3]); 
            M = atoi(argv[4]); 
        }

        if (error) {
            perror("Valor de argumento invalido. ");
            exit(-1);
        }
    }
}

char generar_elemento() {
    if (dificultad) {
        return  (rand() % (6 - 1 + 1) + 1) + '0';
    }
    else {
        return  (rand() % (4 - 1 + 1) + 1) + '0';
    }
    
}

void mostrar_tablero(char* tablero, int n, int m) {
    printf("\nTABLERO: \n"); 
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%2c", tablero[m * i + j]); 
        }
        printf("\n"); 
    }
}

bool pertenece(int* x, int n, int y) {
    bool p = false;
    for (int i = 0; i < n; ++i) {
        p = p || x[i] == y;
    }
    return p;
}

void buscar_camino(char* tablero, int inicio, int fin, int* visitados, int x, int* camino, int y) {
    if (inicio == fin) {
        //visitados[x] = fin; 
        //camino[y] = fin; 
    }
    else {
        //Encima, debajo, izq, dcha
        //Vecino = -1 --> fuera del tablero
        int vecinos[4] = { inicio - M, inicio + M, inicio - 1, inicio + 1 };
        if (vecinos[0] < 0) {
            vecinos[0] = -1;
        }
        if (vecinos[1] >= N * M) {
            vecinos[1] = -1;
        }
        if (inicio % M == 0) {
            vecinos[2] = -1;
        }
        if ((inicio + 1) % M == 0) {
            vecinos[3] = -1;
        }

        for (int i = 0; i < 4; ++i) {
            if (!pertenece(visitados, x, vecinos[i])) {
                if (vecinos[i] != -1) {
                    //Si es vecino se marca como explorado
                    visitados[x] = vecinos[i];
                    ++x;

                    if (tablero[inicio] == tablero[vecinos[i]]) {
                        //En caso de que el vecino sea del mismo tipo, sigo el camino
                        camino[y] = vecinos[i];
                        ++y;
                        buscar_camino(tablero, vecinos[i], fin, visitados, x, camino, y);
                    }
                }
            }
        }
    }
}


int main(int argc, char* argv[]){

    //cargar_argumentos(argc, argv); 
    int tam_tablero = sizeof(char) * N * M; 
    char* tablero = (char*)malloc(tam_tablero);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            tablero[M * i + j] = generar_elemento();
        }
    }

    int selec;
    int id;

    mostrar_tablero(tablero, N, M); 
    printf("Toca elemento: "); 
    scanf("%d", &selec); 
    printf("\nSimular que soy el hilo: "); 
    scanf("%d", &id); 
    printf("\n"); 


    /*char* d_tablero;
    cudaMalloc((void**)&d_tablero, sizeof(char) * N * M); 
    cudaMemcpy(d_tablero, tablero, sizeof(char) * N * M, cudaMemcpyHostToDevice); 

    dim3 bloque(N, M); 
    encontrar_caminos<<<1, bloque>>>(d_tablero, N, M, 2, 3);*/

    

    //Funcion que busque camino
    int* camino = (int*)malloc(tam_tablero);
    int* visitados = (int*)malloc(tam_tablero);

    if(tablero[id] == tablero[selec])
    buscar_camino(tablero, id, selec, visitados, 0, camino, 0);

    printf("\nCamino desde %d\n: ", id);
    for (int i = 0; i < N * M; ++i) {
        printf("%d, ", camino[i]);
    }

    printf("\nVisitados desde %d\n: ", id);
    for (int i = 0; i < N * M; ++i) {
        printf("%d, ", visitados[i]);
    }

    return 0;
}
