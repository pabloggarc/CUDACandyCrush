#include "hip/hip_runtime.h"

#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hiprand/hiprand_kernel.h>

//Variables generales del juego

int N;
int M;
int vidas = 5; 
int modo;
int dificultad; 
__constant__ int nuevos_caramelos_facil[4] = { 1, 2, 3, 4 };
__constant__ int nuevos_caramelos_dificil[6] = { 1, 2, 3, 4, 5, 6 };

//Funciones auxiliares (DEVICE)

/*
    x: vector de enteros
    n: longitud del vector
    y: elemento a encontrar

    Salida: determina si y pertenece a x

*/

__host__ __device__ bool pertenece(int* x, int n, int y) {
    bool p = false;
    for (int i = 0; i < n; ++i) {
        p = p || x[i] == y;
    }
    return p;
}

/*
    tablero: tablero del juego
    inicio: casilla en desde la que se empieza a buscar
    fin: casilla a la que se quiere llegar
    visitados: casillas por las que se ha intentado ir
    x: longitud de visitados
    camino: casillas que forman parte del camino
    y: longitud de camino
    N y M: dimensiones del tablero

    Salida: camino que lleva de inicio a fin (mediante el puntero a camino)

*/

__device__ void buscar_camino(char* tablero, int inicio, int fin, int* visitados, int* x, int* camino, int* y, int N, int M) {
    if (inicio != fin) {
        //Encima, debajo, izq, dcha ||||| Vecino = -1 --> fuera del tablero
        int vecinos[5] = { inicio, inicio - M, inicio + M, inicio - 1, inicio + 1 };
        if (vecinos[1] < 0) {
            vecinos[1] = -1;
        }
        if (vecinos[2] >= N * M) {
            vecinos[2] = -1;
        }
        if (inicio % M == 0) {
            vecinos[3] = -1;
        }
        if ((inicio + 1) % M == 0) {
            vecinos[4] = -1;
        }

        for (int i = 0; i < 5; ++i) {
            if (!pertenece(visitados, *x, vecinos[i])) {
                if (vecinos[i] != -1) {
                    //Se marca como explorado

                    visitados[*x] = vecinos[i];
                    (*x)++;

                    if (tablero[inicio] == tablero[vecinos[i]]) {
                        //En caso de que el vecino sea del mismo tipo, sigo el camino

                        camino[*y] = vecinos[i];
                        (*y)++;
                        buscar_camino(tablero, vecinos[i], fin, visitados, x, camino, y, N, M);
                    }
                }
            }
        }
    }
}

/*
    tablero: tablero del juego
    N y M: dimensiones del tablero
    fila y columna: del elemento tocado

    Salida: tablero con las posiciones a borrar sustituidas por 'X' (mediante el puntero a tablero)

*/

__global__ void encontrar_caminos(char* tablero, int N, int M, int fila, int columna) {
    int selec = fila * M + columna;
    int id = threadIdx.y * N + threadIdx.x;

    //Funcion que busque camino
    int* camino = (int*)malloc(N * M * sizeof(int));
    int* visitados = (int*)malloc(N * M * sizeof(int));
    int x = 1;
    int y = 1;

    for (int i = 1; i < N * M; ++i) {
        camino[i] = -1;
        visitados[i] = -1;
    }

    camino[0] = id;
    visitados[0] = id;

    if (tablero[selec] == tablero[id]) {
        buscar_camino(tablero, id, selec, visitados, &x, camino, &y, N, M);
    }

    if (pertenece(camino, N * M, selec)) {
        for (int i = 0; i < N * M; ++i) {
            int id_camino = camino[i];
            if (id_camino != -1) {
                tablero[id_camino] = 'X';
            }
        }
    }

    free(visitados);
}

/*
    tablero: tablero general del juego con casillas marcadas para borrar
    N y M: dimensiones del tablero
    dif: dificultad de la partida

    Salida: tablero con casillas borradas, desplazadas, y caramelos nuevos introducidos
            en caso de que corresponda (mediante puntero a tablero)

*/

__global__ void recolocar_tablero(char* tablero, int N, int M, int* dif) {
    int id = threadIdx.y * N + threadIdx.x;
    int X_debajo = 0; 
    int noX_encima = 0; 

    char valor_anterior = tablero[id]; 
    for (int i = threadIdx.x; i < N * M; i += M) {
        if (i < id && tablero[i] != 'X') {
            noX_encima++; 
        }
        if (i > id && tablero[i] == 'X') {
            X_debajo++; 
        }
    }

    __syncthreads();

    if (id + M * X_debajo < N * M && X_debajo > 0 && valor_anterior != 'X') {
        tablero[id + M * X_debajo] = valor_anterior; 
    }
    
    if (valor_anterior == 'X') {
        X_debajo++; 
    }

    if (X_debajo - noX_encima > 0) {
        if (*dif) {
            tablero[id] = nuevos_caramelos_dificil[id % 6] + '0'; 
        }
        else {
            tablero[id] = nuevos_caramelos_facil[id % 4] + '0';
        }
    }

}


/*
//kernel que elimina los caramelos de una fila
__global__ void bomba_fila(char* tablero, int N, int M, int fila) {
    int id = threadIdx.y * N + threadIdx.x;

    if (fila == id)
    {
        tablero[id * M] = 'X';   //hacemos que las posiciones de la fila se hagan X para eliminarse despues
    }
}



//kernel que elimina los caramelos de una columna
__global__ void bomba_columna(char* tablero, int N, int M, int columna) {
    int id = threadIdx.y * N + threadIdx.x;

    if (columna == id)
    {
        tablero[id * N] = 'X';   //hacemos que las posiciones de la columna se hagan X para eliminarse despues
    }
}

//kernel que borra los caramelos de un mismo valor
__global__ void bomba_rompecabezas(char* tablero, int numero, int N, int M, int fila, int columna) {
    int id = threadIdx.y * N + threadIdx.x;

    if (fila == id && columna == id)
    {
        tablero[id * M] = 'X';       //si la posicion que le pasamos es la del rompecabezas, se pone a X para eliminarlo
    }

    if (tablero[id * M] == numero)
    {
        tablero[id * M] = 'X';             //si la posicion es del mismo valor que el rompecabes se pone a X para eliminarlo
    }
}


//kernel que borra los caramelos de forma radial
__global__ void bomba_radio(char* tablero, int numero, int N, int M, int fila, int columna) {
    int id = threadIdx.y * N + threadIdx.x;

    if (fila >= N - 4 && fila <= N + 4 && columna >= M - 4 && columna <= M + 4)
    {
        tablero[id * M] = 'X';            //marca a X todos los elementos a 4 de distancia en las 4 direcciones
    }
}
*/

//Funciones auxiliares (HOST)

void cargar_argumentos(int argc, char* argv[]) {
    if (argc != 5) {
        perror("Se esperaban argumentos -a/-m 1/2 n m. "); 
        exit(-1); 
    }
    else {
        int error = 0; 
        if (!strcmp(argv[1], "-a")) {
            modo = 0; 
        }
        else if (!strcmp(argv[1], "-m")){
            modo = 1; 
        }
        else {
            error = 1; 
        }

        if (!strcmp(argv[2], "1")) {
            dificultad = 0; 
        }
        else if (!strcmp(argv[2], "2")) {
            dificultad = 1; 
        }
        else {
            error = 1; 
        }

        if (atoi(argv[3]) < 1 || atoi(argv[4]) < 1) {
            error = 1; 
        }
        else {
            N = atoi(argv[3]); 
            M = atoi(argv[4]); 
        }

        if (error) {
            perror("Valor de argumento invalido. ");
            exit(-1);
        }
    }
}

char generar_elemento() {
    if (dificultad) {
        return  (rand() % (6 - 1 + 1) + 1) + '0';
    }
    else {
        return  (rand() % (4 - 1 + 1) + 1) + '0';
    }
    
}

void mostrar_tablero(char* tablero, int n, int m) {
    printf("\nTABLERO: \n"); 
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%2c", tablero[m * i + j]); 
        }
        printf("\n"); 
    }
}

//Flujo principal

int main(int argc, char* argv[]){

    cargar_argumentos(argc, argv); 
    int tam_tablero = sizeof(char) * N * M; 
    char* tablero = (char*)malloc(tam_tablero);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            tablero[M * i + j] = generar_elemento();
        }
    }

    int* d_dif;
    hipMalloc((void**)&d_dif, sizeof(int));
    hipMemcpy(d_dif, &dificultad, sizeof(int), hipMemcpyHostToDevice);

    while (vidas > 0) {
        mostrar_tablero(tablero, N, M);

        char* d_tablero;
        
        hipMalloc((void**)&d_tablero, sizeof(char) * N * M);
        
        hipMemcpy(d_tablero, tablero, sizeof(char) * N * M, hipMemcpyHostToDevice);
        

        int fila;
        int col; 

        printf("Selecciona fila y columna de la casilla a eliminar: "); 
        scanf("%d %d", &fila, &col); 

        dim3 bloque(N, M);
        encontrar_caminos <<<1, bloque>>> (d_tablero, N, M, fila, col);
        recolocar_tablero <<<1, bloque>>> (d_tablero, N, M, d_dif); 
        hipMemcpy(tablero, d_tablero, sizeof(char) * N * M, hipMemcpyDeviceToHost); 
        hipFree(d_tablero);
    }
    
    free(tablero); 
    hipFree(d_dif); 

    return 0;
}
