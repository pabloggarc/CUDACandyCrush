#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hiprand/hiprand_kernel.h>

//Variables generales del juego

int N;
int M;
int vidas = 5; 
int modo;
int dificultad; 
__constant__ int nuevos_caramelos_facil[4] = { 1, 2, 3, 4 };
__constant__ int nuevos_caramelos_dificil[6] = { 1, 2, 3, 4, 5, 6 };

//Funciones auxiliares (DEVICE)

/*
    x: vector de enteros
    n: longitud del vector
    y: elemento a encontrar

    Salida: determina si y pertenece a x

*/

__host__ __device__ bool pertenece(int* x, int n, int y) {
    bool p = false;
    for (int i = 0; i < n; ++i) {
        p = p || x[i] == y;
    }
    return p;
}

/*
    tablero: tablero del juego
    inicio: casilla en desde la que se empieza a buscar
    fin: casilla a la que se quiere llegar
    visitados: casillas por las que se ha intentado ir
    x: longitud de visitados
    camino: casillas que forman parte del camino
    y: longitud de camino
    N y M: dimensiones del tablero

    Salida: camino que lleva de inicio a fin (mediante el puntero a camino)

*/

__device__ void buscar_camino(char* tablero, int inicio, int fin, int* visitados, int* x, int* camino, int* y, int N, int M) {
    if (inicio != fin) {
        //Encima, debajo, izq, dcha ||||| Vecino = -1 --> fuera del tablero
        int vecinos[5] = { inicio, inicio - M, inicio + M, inicio - 1, inicio + 1 };
        if (vecinos[1] < 0) {
            vecinos[1] = -1;
        }
        if (vecinos[2] >= N * M) {
            vecinos[2] = -1;
        }
        if (inicio % M == 0) {
            vecinos[3] = -1;
        }
        if ((inicio + 1) % M == 0) {
            vecinos[4] = -1;
        }

        for (int i = 0; i < 5; ++i) {
            if (!pertenece(visitados, *x, vecinos[i])) {
                if (vecinos[i] != -1) {
                    //Se marca como explorado

                    visitados[*x] = vecinos[i];
                    (*x)++;

                    if (tablero[inicio] == tablero[vecinos[i]]) {
                        //En caso de que el vecino sea del mismo tipo, sigo el camino

                        camino[*y] = vecinos[i];
                        (*y)++;
                        buscar_camino(tablero, vecinos[i], fin, visitados, x, camino, y, N, M);
                    }
                }
            }
        }
    }
}

/*
    tablero: tablero del juego
    N y M: dimensiones del tablero
    fila y columna: del elemento tocado

    Salida: tablero con las posiciones a borrar sustituidas por 'X' (mediante el puntero a tablero)

*/

__global__ void encontrar_caminos(char* tablero, int N, int M, int fila, int columna) {
    int selec = fila * M + columna;
    int id = threadIdx.y * N + threadIdx.x;

    //Funcion que busque camino
    int* camino = (int*)malloc(N * M * sizeof(int));
    int* visitados = (int*)malloc(N * M * sizeof(int));
    int x = 1;
    int y = 1;

    for (int i = 1; i < N * M; ++i) {
        camino[i] = -1;
        visitados[i] = -1;
    }

    camino[0] = id;
    visitados[0] = id;

    if (tablero[selec] == tablero[id]) {
        buscar_camino(tablero, id, selec, visitados, &x, camino, &y, N, M);
    }

    if (pertenece(camino, N * M, selec)) {
        for (int i = 0; i < N * M; ++i) {
            int id_camino = camino[i];
            if (id_camino != -1) {
                tablero[id_camino] = 'X';
            }
        }
    }

    free(visitados);
}

/*
    tablero: tablero general del juego con casillas marcadas para borrar
    N y M: dimensiones del tablero
    dif: dificultad de la partida

    Salida: tablero con casillas borradas, desplazadas, y caramelos nuevos introducidos
            en caso de que corresponda (mediante puntero a tablero)

*/

__global__ void recolocar_tablero(char* tablero, int N, int M, int* dif) {
    int id = threadIdx.y * N + threadIdx.x;
    int X_debajo = 0; 
    int noX_encima = 0; 

    char valor_anterior = tablero[id]; 
    for (int i = threadIdx.x; i < N * M; i += M) {
        if (i < id && tablero[i] != 'X') {
            noX_encima++; 
        }
        if (i > id && tablero[i] == 'X') {
            X_debajo++; 
        }
    }

    __syncthreads();

    if (id + M * X_debajo < N * M && X_debajo > 0 && valor_anterior != 'X') {
        tablero[id + M * X_debajo] = valor_anterior; 
    }
    
    if (valor_anterior == 'X') {
        X_debajo++; 
    }

    if (X_debajo - noX_encima > 0) {
        if (*dif) {
            tablero[id] = nuevos_caramelos_dificil[id % 6] + '0'; 
        }
        else {
            tablero[id] = nuevos_caramelos_facil[id % 4] + '0';
        }
    }

}


/*void bloquesEspeciales(char* tablero, int N, int M, int fila, int columna, int longitud) //longitud del camino
{
    int indiceSeleccionado = fila * M + columna;

    if(longitud == 5){ // Si hay cinco bloques del mismo color, uno al lado del otro, cuando lo toca obtiene una bomba. Cuando
                        //lo toca, borra todos los bloques en esa fila o columna de forma aleatoria.
        //genero bloque BOMBA
        tablero[indiceSeleccionado] = 'B';

    }
    else if(longitud == 6){
        //genero bloque TNT
        tablero[indiceSeleccionado] = 'T';
    }
    else if(longitud >= 7){
        //genero bloque ROMPECABEZAS
        tablero[indiceSeleccionado] = 'R';
    }
}*/

//Funciones auxiliares (HOST)

void cargar_argumentos(int argc, char* argv[]) {
    if (argc != 5) {
        perror("Se esperaban argumentos -a/-m 1/2 n m. "); 
        exit(-1); 
    }
    else {
        int error = 0; 
        if (!strcmp(argv[1], "-a")) {
            modo = 0; 
        }
        else if (!strcmp(argv[1], "-m")){
            modo = 1; 
        }
        else {
            error = 1; 
        }

        if (!strcmp(argv[2], "1")) {
            dificultad = 0; 
        }
        else if (!strcmp(argv[2], "2")) {
            dificultad = 1; 
        }
        else {
            error = 1; 
        }

        if (atoi(argv[3]) < 1 || atoi(argv[4]) < 1) {
            error = 1; 
        }
        else {
            N = atoi(argv[3]); 
            M = atoi(argv[4]); 
        }

        if (error) {
            perror("Valor de argumento invalido. ");
            exit(-1);
        }
    }
}

char generar_elemento() {
    if (dificultad) {
        return  (rand() % (6 - 1 + 1) + 1) + '0';
    }
    else {
        return  (rand() % (4 - 1 + 1) + 1) + '0';
    }
    
}

void mostrar_tablero(char* tablero, int n, int m) {
    printf("\nTABLERO: \n"); 
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%2c", tablero[m * i + j]); 
        }
        printf("\n"); 
    }
}

//Flujo principal

int main(int argc, char* argv[]) {

    srand(time(NULL)); //semilla para la ejecucion automatica
    cargar_argumentos(argc, argv); //aqui ya que es N y M
    int tam_tablero = sizeof(char) * N * M;
    char* tablero = (char*)malloc(tam_tablero);
    int cuantos_hay = 0; //esta en host
    int posicion = 0; //esta en host


    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            tablero[M * i + j] = generar_elemento();
        }
    }

    int* d_dif;
    hipMalloc((void**)&d_dif, sizeof(int));
    hipMemcpy(d_dif, &dificultad, sizeof(int), hipMemcpyHostToDevice);

    while (vidas > 0) {
        mostrar_tablero(tablero, N, M);

        char* d_tablero;

        hipMalloc((void**)&d_tablero, sizeof(char) * N * M);

        hipMemcpy(d_tablero, tablero, sizeof(char) * N * M, hipMemcpyHostToDevice);


        int fila;
        int col;
        //Pedir fila y columna al usuario
        if (modo == 1)
        {
            //Ejecucion manual
            printf("Selecciona fila y columna de la casilla a eliminar: ");
            scanf("%d %d", &fila, &col);
            //Comprobar que la fila y columna son validas
            while (fila < 0 || fila >= N || col < 0 || col >= M)
            {
                printf("Introduce una fila y columna validas:\n");
                printf("Selecciona fila y columna de la casilla a eliminar: ");
                scanf("%d %d", &fila, &col);
            }
        }
        else
        {
            //Ejecucion automatica

            //Generar fila y columna aleatorias
            fila = rand() % N;
            col = rand() % M;
            printf("Seleccionada fila %d y columna %d\n", fila, col);
        }

       
        

        dim3 bloque(N, M);
        encontrar_caminos << <1, bloque >> > (d_tablero, N, M, fila, col); //genera camino
        hipMemcpy(tablero, d_tablero, sizeof(char) * N * M, hipMemcpyDeviceToHost);

        //voy a contar las X del tablero que me ha retornado el kernel encontrar_caminos
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < M; j++) {
                if (tablero[M * i + j] == 'X') { 
                    cuantos_hay++;
                   
                }
            }
        }

        //obtengo el valor de la posicion seleccionada
        posicion = tablero[M * fila + col];


        printf("\nTermino de cambiar los valores por X\n");
        mostrar_tablero(tablero, N, M);



        //si cuantos_hay es 1 (solo hay una X) y por tanto se resta la vida porque no puede formar caminos:
        if (cuantos_hay == 1)
        {
            //resto una vida
            vidas--;
            printf("\nNo hay caminos, pierdes una vida. Te quedan %d vidas.\n", vidas);
            mostrar_tablero(tablero, N, M);
        }

        if (cuantos_hay == 5) { // Si hay cinco bloques del mismo color, uno al lado del otro, cuando lo toca obtiene una bomba. Cuando 
                            //lo toca, borra todos los bloques en esa fila o columna de forma aleatoria.
            //genero bloque BOMBA

            posicion = 'B';
            printf("entro en el if del bloque bomba");
            mostrar_tablero(tablero, N, M);
        }

        if (cuantos_hay == 6) {
            //genero bloque TNT
            posicion = 'T';
            printf("entro en el if del bloque tnt");
            mostrar_tablero(tablero, N, M);
        }

        if (cuantos_hay >= 7) {
            //genero bloque ROMPECABEZAS
            posicion = 'R';
            printf("entro en el if del bloque de rafael rico");
            mostrar_tablero(tablero, N, M);
        }

        printf("\nsi hay camino, print para debuggear");
        recolocar_tablero << <1, bloque >> > (d_tablero, N, M, d_dif); //borra camino
        hipMemcpy(tablero, d_tablero, sizeof(char) * N * M, hipMemcpyDeviceToHost);
        hipFree(d_tablero);
        cuantos_hay = 0; //reinicio el contador de Xs

        //imprimo las vidas 
        printf("Vidas: %d", vidas);
    }
    printf("\nFIN DEL JUEGO, TE HAS QUEDADO SIN VIDAS, mira wasap");

    free(tablero);
    hipFree(d_dif);

    return 0;
}
