#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

//Variables generales del juego

int N;
int M;
int vidas = 5; 
int modo;
int dificultad; 
__constant__ int nuevos_caramelos_facil[4] = { 1, 2, 3, 4 };
__constant__ int nuevos_caramelos_dificil[6] = { 1, 2, 3, 4, 5, 6 };

//Funciones auxiliares (DEVICE)

/*
    x: vector de enteros
    n: longitud del vector
    y: elemento a encontrar

    Salida: determina si y pertenece a x

*/

__host__ __device__ bool pertenece(int* x, int n, int y) {
    bool p = false;
    for (int i = 0; i < n; ++i) {
        p = p || x[i] == y;
    }
    return p;
}

/*
    tablero: tablero del juego
    inicio: casilla en desde la que se empieza a buscar
    fin: casilla a la que se quiere llegar
    visitados: casillas por las que se ha intentado ir
    x: longitud de visitados
    camino: casillas que forman parte del camino
    y: longitud de camino
    N y M: dimensiones del tablero

    Salida: camino que lleva de inicio a fin (mediante el puntero a camino)

*/

__device__ void buscar_camino(char* tablero, int inicio, int fin, int* visitados, int* x, int* camino, int* y, int N, int M) {
    if (inicio != fin) {
        //Encima, debajo, izq, dcha ||||| Vecino = -1 --> fuera del tablero
        int vecinos[5] = { inicio, inicio - M, inicio + M, inicio - 1, inicio + 1 };
        if (vecinos[1] < 0) {
            vecinos[1] = -1;
        }
        if (vecinos[2] >= N * M) {
            vecinos[2] = -1;
        }
        if (inicio % M == 0) {
            vecinos[3] = -1;
        }
        if ((inicio + 1) % M == 0) {
            vecinos[4] = -1;
        }

        for (int i = 0; i < 5; ++i) {
            if (!pertenece(visitados, *x, vecinos[i])) {
                if (vecinos[i] != -1) {
                    //Se marca como explorado

                    visitados[*x] = vecinos[i];
                    (*x)++;

                    if (tablero[inicio] == tablero[vecinos[i]]) {
                        //En caso de que el vecino sea del mismo tipo, sigo el camino

                        camino[*y] = vecinos[i];
                        (*y)++;
                        buscar_camino(tablero, vecinos[i], fin, visitados, x, camino, y, N, M);
                    }
                }
            }
        }
    }
}

/*
    tablero: tablero del juego
    N y M: dimensiones del tablero
    fila y columna: del elemento tocado

    Salida: tablero con las posiciones a borrar sustituidas por 'X' (mediante el puntero a tablero)

*/

__global__ void encontrar_caminos(char* tablero, int N, int M, int fila, int columna, int* borrados) {
    int selec = fila * M + columna;
    int id = threadIdx.y * N + threadIdx.x;

    //Funcion que busque camino
    int* camino = (int*)malloc(N * M * sizeof(int));
    int* visitados = (int*)malloc(N * M * sizeof(int));
    int x = 1;
    int y = 1;

    for (int i = 1; i < N * M; ++i) {
        camino[i] = -1;
        visitados[i] = -1;
    }

    camino[0] = id;
    visitados[0] = id;

    if (tablero[selec] == tablero[id]) {
        buscar_camino(tablero, id, selec, visitados, &x, camino, &y, N, M);
    }

    if (pertenece(camino, N * M, selec) && x > 1) {
        for (int i = 0; i < N * M; ++i) {
            int id_camino = camino[i];
            if (id_camino != -1) {
                tablero[id_camino] = 'X';
            }
        }
    }

    if (tablero[id] == 'X') {
        atomicAdd(borrados, 1);
    }

    free(visitados);
}

/*
    tablero: tablero general del juego con casillas marcadas para borrar
    N y M: dimensiones del tablero
    dif: dificultad de la partida

    Salida: tablero con casillas borradas, desplazadas, y caramelos nuevos introducidos
            en caso de que corresponda (mediante puntero a tablero)

*/

__global__ void recolocar_tablero(char* tablero, int N, int M, int* dif) {
    int id = threadIdx.y * N + threadIdx.x;
    int X_debajo = 0; 
    int noX_encima = 0; 

    char valor_anterior = tablero[id]; 
    for (int i = threadIdx.x; i < N * M; i += M) {
        if (i < id && tablero[i] != 'X') {
            noX_encima++; 
        }
        if (i > id && tablero[i] == 'X') {
            X_debajo++; 
        }
    }

    __syncthreads();

    if (id + M * X_debajo < N * M && X_debajo > 0 && valor_anterior != 'X') {
        tablero[id + M * X_debajo] = valor_anterior; 
    }
    
    if (valor_anterior == 'X') {
        X_debajo++; 
    }

    if (X_debajo - noX_encima > 0) {
        if (*dif) {
            tablero[id] = nuevos_caramelos_dificil[id % 6] + '0'; 
        }
        else {
            tablero[id] = nuevos_caramelos_facil[id % 4] + '0';
        }
    }

}


__global__ void bloquesEspeciales(char* tablero, int N, int M, int fila, int columna, int* borrados) {

    int id = threadIdx.y * N + threadIdx.x;
    int seleccionado = fila * M + columna; 

    if (tablero[seleccionado] == 'B'){

        //Borro fila o columna de forma aleatoria

        int borradoAleatorio = clock64() % 2; 

        if ((borradoAleatorio && threadIdx.x == columna) ||
            (borradoAleatorio && threadIdx.y == fila)){

            tablero[id] = 'X'; 
            atomicAdd(borrados, 1);

            printf("Ha explotado la bomba (%d, %d)\n", fila, columna);
        }
    }

    if (tablero[seleccionado] == 'T'){
        //Borro todo en un radio de 4 desde el elemento seleccionado
        
        if (((threadIdx.x + 4 == columna) || (threadIdx.x - 4 == columna)) &&
            ((threadIdx.y + 4 == fila) || (threadIdx.y - 4 == fila))) {

            tablero[id] = 'X'; 
            atomicAdd(borrados, 1);

            printf("Se ha aplicado el efecto del TNT (%d, %d)\n", fila, columna);
        }
    }

    if (tablero[seleccionado] == 'R'){
        //Borro todos los elementos del tipo
        
        char tipo = clock64() % 6 + 1 + '0';
        if (tablero[seleccionado] == tipo) {
            tablero[id] = 'X'; 
            atomicAdd(borrados, 1);

            printf("Se ha aplicado el efecto del rompecabezas (%d, %d)\n", fila, columna);
        }
    }
    
}

//Funciones auxiliares (HOST)

void cargar_argumentos(int argc, char* argv[]) {
    if (argc != 5) {
        perror("Se esperaban argumentos -a/-m 1/2 n m. "); 
        exit(-1); 
    }
    else {
        int error = 0; 
        if (!strcmp(argv[1], "-a")) {
            modo = 0; 
        }
        else if (!strcmp(argv[1], "-m")){
            modo = 1; 
        }
        else {
            error = 1; 
        }

        if (!strcmp(argv[2], "1")) {
            dificultad = 0; 
        }
        else if (!strcmp(argv[2], "2")) {
            dificultad = 1; 
        }
        else {
            error = 1; 
        }

        if (atoi(argv[3]) < 1 || atoi(argv[4]) < 1) {
            error = 1; 
        }
        else {
            N = atoi(argv[3]); 
            M = atoi(argv[4]); 
        }

        if (error) {
            perror("Valor de argumento invalido. ");
            exit(-1);
        }
    }
}

char generar_elemento() {
    if (dificultad) {
        return  (rand() % (6 - 1 + 1) + 1) + '0';
    }
    else {
        return  (rand() % (4 - 1 + 1) + 1) + '0';
    }
    
}

void mostrar_tablero(char* tablero, int n, int m) {
    printf("\nTABLERO: \n"); 
    for (int i = 0; i < n; i++) {
        printf("\t\t\t\t"); 
        for (int j = 0; j < m; j++) {
            printf("%2c", tablero[m * i + j]); 
        }
        printf("\n"); 
    }
    printf("\n"); 
}

//Flujo principal

int main(int argc, char* argv[]) {
    srand(time(NULL)); //semilla para la ejecucion automatica
    cargar_argumentos(argc, argv); //aqui ya que es N y M
    int tam_tablero = sizeof(char) * N * M;
    char* tablero = (char*)malloc(tam_tablero);
    int posicion = 0; //esta en host

    printf("n, m = %d, %d", N, M); 

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            tablero[M * i + j] = generar_elemento();
        }
    }

    int* d_dif;
    hipMalloc((void**)&d_dif, sizeof(int));
    hipMemcpy(d_dif, &dificultad, sizeof(int), hipMemcpyHostToDevice);

    while (vidas > 0) {
        mostrar_tablero(tablero, N, M);

        char* d_tablero;
        int* d_X; 

        hipMalloc((void**)&d_tablero, sizeof(char) * N * M);
        hipMalloc((void**)&d_X, sizeof(int)); 
        hipMemcpy(d_tablero, tablero, sizeof(char) * N * M, hipMemcpyHostToDevice);


        int fila;
        int col;
        //Pedir fila y columna al usuario
        if (modo == 1){

            //Ejecucion manual
            printf("Selecciona fila y columna de la casilla a eliminar: ");
            scanf("%d %d", &fila, &col);

            //Comprobar que la fila y columna son validas
            while (fila < 0 || fila >= N || col < 0 || col >= M){
                printf("Introduce una fila y columna validas:\n");
                printf("Selecciona fila y columna de la casilla a eliminar: ");
                scanf("%d %d", &fila, &col);
            }
        }
        else{
            //Ejecucion automatica

            //Generar fila y columna aleatorias
            fila = rand() % N;
            col = rand() % M;
            printf("Seleccionada fila %d y columna %d\n", fila, col);
        }

        int seleccionado = fila * M + col;
        int borrados = 0; 
        dim3 bloque(N, M); 

        //Comprobar si el elemento seleccionado es un n�mero

        if (tablero[seleccionado] >= 49 && tablero[seleccionado] <= 54){
            encontrar_caminos <<<1, bloque>>> (d_tablero, N, M, fila, col, d_X); 
        } 
        else{
            bloquesEspeciales<<<1, bloque>>>(tablero, N, M, fila, col, d_X);
        }

        hipMemcpy(tablero, d_tablero, sizeof(char) * N * M, hipMemcpyDeviceToHost);
        hipMemcpy(&borrados, d_X, sizeof(int), hipMemcpyDeviceToHost);
        
        //Decidimos qu� pasa en funci�n de los que se han borrado

        printf("debug>BORRADOS: %d\n", borrados); 
        
        if (borrados == 0){
            vidas--;
            printf("\nNo hay suficientes caramelos juntos, pierdes una vida!\n");
        }
        else if (borrados == 5) {
            tablero[seleccionado] = 'B';
        }
        else if (borrados == 6) {
            tablero[seleccionado] = 'T';
        }
        else if (borrados >= 7) {
            tablero[seleccionado] = 'R';
        }

        mostrar_tablero(tablero, N, M); 
        hipMemcpy(d_tablero, tablero, sizeof(char) * N * M, hipMemcpyHostToDevice);

        //Bajamos caramelos y metemos nuevos

        recolocar_tablero << <1, bloque >> > (d_tablero, N, M, d_dif); 
        hipMemcpy(tablero, d_tablero, sizeof(char) * N * M, hipMemcpyDeviceToHost);
        hipFree(d_tablero);

        printf("Vidas: %d\n", vidas);
    }
    printf("\nFIN DEL JUEGO, TE HAS QUEDADO SIN VIDAS");

    free(tablero);
    hipFree(d_dif);

    return 0;
}
