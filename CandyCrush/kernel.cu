#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

int N = 1; //filas
int M = 1;  //columnas
int vidas = 5; 
int dificultad; 
int modo; 

__host__ __device__ bool pertenece(int* x, int n, int y) {
    bool p = false;
    for (int i = 0; i < n; ++i) {
        p = p || x[i] == y;
    }
    return p;
}

__device__ void buscar_camino(char* tablero, int inicio, int fin, int* visitados, int* x, int* camino, int* y, int N, int M) {
    if (inicio != fin) {
        //Encima, debajo, izq, dcha ||||| Vecino = -1 --> fuera del tablero
        int vecinos[5] = { inicio, inicio - M, inicio + M, inicio - 1, inicio + 1 };
        if (vecinos[1] < 0) {
            vecinos[1] = -1;
        }
        if (vecinos[2] >= N * M) {
            vecinos[2] = -1;
        }
        if (inicio % M == 0) {
            vecinos[3] = -1;
        }
        if ((inicio + 1) % M == 0) {
            vecinos[4] = -1;
        }

        for (int i = 0; i < 5; ++i) {
            if (!pertenece(visitados, *x, vecinos[i])) {
                if (vecinos[i] != -1) {
                    //Se marca como explorado

                    visitados[*x] = vecinos[i];
                    (*x)++;

                    if (tablero[inicio] == tablero[vecinos[i]]) {
                        //En caso de que el vecino sea del mismo tipo, sigo el camino

                        camino[*y] = vecinos[i];
                        (*y)++;
                        buscar_camino(tablero, vecinos[i], fin, visitados, x, camino, y, N, M);
                    }
                }
            }
        }
    }
}


/*
//kernel que elimina los caramelos de una fila
__global__ void bomba_fila(char* tablero, int N, int M, int fila) {
    int id = threadIdx.y * N + threadIdx.x;

    if (fila == id)
    {
        tablero[id * M] = 'X';   //hacemos que las posiciones de la fila se hagan X para eliminarse despues
    }
}



//kernel que elimina los caramelos de una columna
__global__ void bomba_columna(char* tablero, int N, int M, int columna) {
    int id = threadIdx.y * N + threadIdx.x;

    if (columna == id)
    {
        tablero[id * N] = 'X';   //hacemos que las posiciones de la columna se hagan X para eliminarse despues
    }
}

//kernel que borra los caramelos de un mismo valor
__global__ void bomba_rompecabezas(char* tablero, int numero, int N, int M, int fila, int columna) {
    int id = threadIdx.y * N + threadIdx.x;

    if (fila == id && columna == id)
    {
        tablero[id * M] = 'X';       //si la posicion que le pasamos es la del rompecabezas, se pone a X para eliminarlo
    }

    if (tablero[id * M] == numero)
    {
        tablero[id * M] = 'X';             //si la posicion es del mismo valor que el rompecabes se pone a X para eliminarlo
    }
}


//kernel que borra los caramelos de forma radial
__global__ void bomba_radio(char* tablero, int numero, int N, int M, int fila, int columna) {
    int id = threadIdx.y * N + threadIdx.x;

    if (fila >= N - 4 && fila <= N + 4 && columna >= M - 4 && columna <= M + 4)
    {
        tablero[id * M] = 'X';            //marca a X todos los elementos a 4 de distancia en las 4 direcciones
    }
}
*/


__global__ void encontrar_caminos(char* tablero, int N, int M, int fila, int columna) {
    int selec = fila * M + columna; 
    int id = threadIdx.y * N + threadIdx.x; 

    //Funcion que busque camino
    int* camino = (int*)malloc(N * M * sizeof(int)); 
    int* visitados = (int*)malloc(N * M * sizeof(int));
    int x = 1; 
    int y = 1; 

    for (int i = 1; i < N * M; ++i) {
        camino[i] = -1; 
        visitados[i] = -1; 
    }

    camino[0] = id; 
    visitados[0] = id; 

    if (tablero[selec] == tablero[id]) {
        buscar_camino(tablero, id, selec, visitados, &x, camino, &y, N, M);
    }

    if (pertenece(camino, N * M, selec)) {
        for (int i = 0; i < N * M; ++i) {
            int id_camino = camino[i];
            if (id_camino != -1) {
                tablero[id_camino] = 'X';
            }
        }
    }

    
    free(visitados); 
}

void cargar_argumentos(int argc, char* argv[]) {
    if (argc != 5) {
        perror("Se esperaban argumentos -a/-m 1/2 n m. "); 
        exit(-1); 
    }
    else {
        int error = 0; 
        if (!strcmp(argv[1], "-a")) {
            modo = 0; 
        }
        else if (!strcmp(argv[1], "-m")){
            modo = 1; 
        }
        else {
            error = 1; 
        }

        if (!strcmp(argv[2], "1")) {
            dificultad = 0; 
        }
        else if (!strcmp(argv[2], "2")) {
            dificultad = 1; 
        }
        else {
            error = 1; 
        }

        if (atoi(argv[3]) < 1 || atoi(argv[4]) < 1) {
            error = 1; 
        }
        else {
            N = atoi(argv[3]); 
            M = atoi(argv[4]); 
        }

        if (error) {
            perror("Valor de argumento invalido. ");
            exit(-1);
        }
    }
}

char generar_elemento() {
    if (dificultad) {
        return  (rand() % (6 - 1 + 1) + 1) + '0';
    }
    else {
        return  (rand() % (4 - 1 + 1) + 1) + '0';
    }
    
}

void mostrar_tablero(char* tablero, int n, int m) {
    printf("\nTABLERO: \n"); 
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%2c", tablero[m * i + j]); 
        }
        printf("\n"); 
    }
}


int main(int argc, char* argv[]){

    cargar_argumentos(argc, argv); 
    int tam_tablero = sizeof(char) * N * M; 
    char* tablero = (char*)malloc(tam_tablero);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            tablero[M * i + j] = generar_elemento();
        }
    }


    while (vidas > 0) {
        mostrar_tablero(tablero, N, M);

        char* d_tablero;
        hipMalloc((void**)&d_tablero, sizeof(char) * N * M);
        hipMemcpy(d_tablero, tablero, sizeof(char) * N * M, hipMemcpyHostToDevice);

        int fila;
        int col; 

        printf("Selecciona fila y columna de la casilla a eliminar: "); 
        scanf("%d %d", &fila, &col); 

        dim3 bloque(N, M);
        encontrar_caminos <<<1, bloque >>> (d_tablero, N, M, fila, col);
        hipMemcpy(tablero, d_tablero, sizeof(char) * N * M, hipMemcpyDeviceToHost); 
    }

    
    
    free(tablero); 

    return 0;
}
